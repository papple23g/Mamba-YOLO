/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/
#ifndef M_LOG2E
#define M_LOG2E 1.4426950408889634074
#endif

#include "selective_scan_bwd_kernel_ndstate.cuh"

template void selective_scan_bwd_cuda<1, float, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<1, at::Half, float>(SSMParamsBwd &params, hipStream_t stream);
template void selective_scan_bwd_cuda<1, at::BFloat16, float>(SSMParamsBwd &params, hipStream_t stream);

